#include "hip/hip_runtime.h"
#include "CUDAfunctions.cuh"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string>
#include "Spinnaker.h"
#include "SpinGenApi/SpinnakerGenApi.h"
#include <unistd.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core/types_c.h>

int main(int argc, char** argv){
    std::cout << argv[0] << " Starting..." << std::endl;
    
    // Parameters
    const float camExposure = 800.0;
    const float camExpRatio = 0.78;

    const int OffsetX = atoi(argv[1]);
    // const int OffsetX = 584;
    const int OffsetY = atoi(argv[2]);
    // const int OffsetY = 506;
    
    const int imgLen = 512;
    const int intrSize = imgLen/8;
    const int gridSize = imgLen/8;
    const int srchSize = imgLen/4;
    const int gridNum = (int)(imgLen/gridSize);

    const float zFront = 1000*60.0;
    const float dz = 50.0;
    const float wavLen = 0.532;
    const float dx = 3.45/0.5;

    const int blockSize = 16; 


    // Camera Init
    Spinnaker::SystemPtr system = Spinnaker::System::GetInstance();
    Spinnaker::CameraList camList = system->GetCameras();
    unsigned int numCameras = camList.GetSize();
    if (numCameras==0){
        std::cout << "No Cameras are Connected! Quitting..." << std::endl;
        exit(1);
    }
    Spinnaker::CameraPtr pCam[numCameras];
    std::cout << "Camera" << "\t" << "ModelName" << "\t\t\t" << "SerialNumber" << std::endl;
    for (int i = 0; i < numCameras; i++){
        pCam[i] = camList.GetByIndex(i);
        pCam[i]->Init();
        Spinnaker::GenICam::gcstring modelName = pCam[i]->TLDevice.DeviceModelName.GetValue();
        Spinnaker::GenICam::gcstring serialNum = pCam[i]->TLDevice.DeviceSerialNumber.GetValue();
        std::cout << i << "\t" << modelName << "\t" << serialNum << std::endl;
    }
    if (numCameras != 2){
        std::cout << "Number of Connected Cameras is not 2. Quitting..." << std::endl;
        exit(0);
    }
    std::cout << "Camera Enum OK" << std::endl;

    // Camera Setup
    cameraSetup(pCam,imgLen,OffsetX,OffsetY,camExposure,camExpRatio);

    float mean1, mean2, exp1, exp2;
    exp1 = pCam[0]->ExposureTime.GetValue();
    exp2 = pCam[1]->ExposureTime.GetValue();
    // Processing
    while(1){
        std::tie(mean1,mean2) = getCamMean(pCam,imgLen);
        if (abs(mean1-0.5)<=0.01 && abs(mean2-0.5) <= 0.01){
            break;
        }else if(abs(mean2-0.5)<=0.01){
            exp1 += (mean1-0.5)*100.0;
            pCam[0]->ExposureTime.SetValue((double)exp1);
        }else{
            exp2 += (mean2-0.5)*100.0;
            pCam[1]->ExposureTime.SetValue((double)exp2);
        }

    }
    exp1 = pCam[0]->ExposureTime.GetValue();
    exp2 = pCam[1]->ExposureTime.GetValue();
    std::cout << "Cam1 Exposure:" << exp1 << std::endl;
    std::cout << "Cam2 Exposure:" << exp2 << std::endl;
    std::cout << "Exp ratio: " << exp2/exp1 << std::endl;
    
    camList.Clear();
    system->ReleaseInstance();

    hipDeviceReset();
    return 0;
}