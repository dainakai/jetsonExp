#include "CUDAfunctions.cuh"
// #include "hostfunctions.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string>
#include "Spinnaker.h"
#include "SpinGenApi/SpinnakerGenApi.h"
#include <unistd.h>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core/types_c.h>

int main(int argc, char** argv){    
    // Parameters
    const float camExposure = 700.0;
    const float camExpRatio = 0.78;

    const int OffsetX = atoi(argv[1]);
    // const int OffsetX = 592;
    const int OffsetY = atoi(argv[2]);
    // const int OffsetY = 510;
    
    const int imgLen = 512;
    const int intrSize = imgLen/8;
    const int gridSize = imgLen/8;
    const int srchSize = imgLen/4;
    const int gridNum = (int)(imgLen/gridSize);

    const float zFront = 1000*60.0;
    const float dz = 50.0;
    const float wavLen = 0.532;
    const float dx = 3.45/0.5;

    const int blockSize = 16; 


    // Camera Init
    Spinnaker::SystemPtr system = Spinnaker::System::GetInstance();
    Spinnaker::CameraList camList = system->GetCameras();
    unsigned int numCameras = camList.GetSize();
    if (numCameras==0){
        std::cout << "No Cameras are Connected! Quitting..." << std::endl;
        exit(1);
    }
    Spinnaker::CameraPtr pCam[numCameras];
    std::cout << "Camera" << "\t" << "ModelName" << "\t\t\t" << "SerialNumber" << std::endl;
    for (int i = 0; i < numCameras; i++){
        pCam[i] = camList.GetByIndex(i);
        pCam[i]->Init();
        Spinnaker::GenICam::gcstring modelName = pCam[i]->TLDevice.DeviceModelName.GetValue();
        Spinnaker::GenICam::gcstring serialNum = pCam[i]->TLDevice.DeviceSerialNumber.GetValue();
        std::cout << i << "\t" << modelName << "\t" << serialNum << std::endl;
    }
    if (numCameras != 2){
        std::cout << "Number of Connected Cameras is not 2. Quitting..." << std::endl;
        exit(0);
    }
    std::cout << "Camera Enum OK" << std::endl;

    cameraSetup(pCam,imgLen,OffsetX,OffsetY,camExposure,camExpRatio);

    getImgAndBundleAdjCheck(pCam,imgLen,gridSize,intrSize,srchSize,zFront,dz,wavLen,dx,blockSize);

    
    pCam[0]->DeInit();
    pCam[1]->DeInit();
    system->ReleaseInstance();

    hipDeviceReset();
    return 0;
}