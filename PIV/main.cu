#include "CUDAfunctions.cuh"
#include "hostfunctions.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "Spinnaker.h"
#include "SpinGenApi/SpinnakerGenApi.h"
// #include <opencv2/core.hpp>
// #include <opencv2/imgcodecs.hpp>
// #include <opencv2/highgui.hpp>
// #include <opencv2/core/types_c.h>
// #include <gtk/gtk.h>

int main(int argc, char** argv){
    std::cout << argv[0] << " Starting..." << std::endl;
    // printf("%s Starting...\n", argv[0]);

    // const char* pathImg1 = "./cam1.bmp";
    // const char* pathImg2 = "./cam2.bmp";

    // const int imgLen = 1024;
    // const int intrSize = imgLen/8;
    // const int gridSize = imgLen/8;
    // const int srchSize = imgLen/4;
    // const int gridNum = (int)(imgLen/gridSize);

    // const int blockSize = 16;

    // unsigned char *UIntimage1, *UIntimage2;
    // float *fimg1, *fimg2;
    // UIntimage1 = (unsigned char*)malloc(sizeof(unsigned char)*imgLen*imgLen);
    // UIntimage2 = (unsigned char*)malloc(sizeof(unsigned char)*imgLen*imgLen);
    // fimg1 = (float*)malloc(sizeof(float)*imgLen*imgLen);
    // fimg2 = (float*)malloc(sizeof(float)*imgLen*imgLen);

    // getFloatimage(fimg1,imgLen,pathImg1);
    // getFloatimage(fimg2,imgLen,pathImg2);

    // float vecArrayX[(gridNum-1)*(gridNum-1)];
    // float vecArrayY[(gridNum-1)*(gridNum-1)];
    // float *pvecArrX = (float *)vecArrayX;
    // float *pvecArrY = (float *)vecArrayY;
    // getPIVMapOnGPU(pvecArrX,pvecArrY,fimg1,fimg2,imgLen,gridSize,intrSize,srchSize,blockSize);

    // for (int i = 0; i < gridNum-1; i++)
    // {
    //     for (int j = 0; j < gridNum-1; j++)
    //     {
    //         printf("vx: %f\t vy: %f\n",vecArrayX[i*(gridNum-1) +j],vecArrayY[i*(gridNum-1) +j]);
    //     }
    //     printf("\n");
    // }

    // saveVecArray(vecArrayX,vecArrayY,gridSize,gridNum);
    // plotVecFieldOnGnuplot(imgLen);
    
    // Parameters
    const float camExposure = 400.0;
    const float camGain = 0.0;
    
    const int imgLen = 1024;
    const int intrSize = imgLen/8;
    const int gridSize = imgLen/8;
    const int srchSize = imgLen/4;
    const int gridNum = (int)(imgLen/gridSize);

    const float zFront = 1000*100.0;
    const float dz = 50.0;
    const float wavLen = 0.532;
    const float dx = 3.45/0.5;

    const int blockSize = 16; 


    // Camera Init
    Spinnaker::SystemPtr system = Spinnaker::System::GetInstance();
    Spinnaker::CameraList camList = system->GetCameras();
    unsigned int numCameras = camList.GetSize();
    if (numCameras==0){
        // printf("No Cameras are Connected! Quitting...\n");
        std::cout << "No Cameras are Connected! Quitting..." << std::endl;
        exit(1);
    }
    Spinnaker::CameraPtr pCam[numCameras];
    std::cout << "Camera" << "\t" << "ModelName" << "\t\t\t" << "SerialNumber" << std::endl;
    for (int i = 0; i < numCameras; i++){
        pCam[i] = camList.GetByIndex(i);
        pCam[i]->Init();
        Spinnaker::GenICam::gcstring modelName = pCam[i]->TLDevice.DeviceModelName.GetValue();
        Spinnaker::GenICam::gcstring serialNum = pCam[i]->TLDevice.DeviceSerialNumber.GetValue();
        // printf("%d\t%s\t%s\n",i,modelName,serialNum);
        std::cout << i << "\t" << modelName << "\t" << serialNum << std::endl;
    }
    if (numCameras != 2){
        // printf("Number of Connected Cameras is not 2. Quitting...\n");
        std::cout << "Number of Connected Cameras is not 2. Quitting..." << std::endl;
        exit(0);
    }
    // printf("\n");
    std::cout << "Camera Enum OK" << std::endl;

    cameraSetup(pCam,1024,100,100);
    getImgAndPIV(pCam,imgLen,gridSize,intrSize,srchSize,zFront,dz,wavLen,dx,blockSize);
    
    pCam[0]->DeInit();
    pCam[1]->DeInit();
    system->ReleaseInstance();

    // GtkWidget* window;
    // gtk_init(&argc,&argv);
    // window = gtk_window_new(GTK_WINDOW_TOPLEVEL);
    // gtk_widget_set_size_request(window,100,50);
    // gtk_window_set_title(GTK_WINDOW(window),"Controller");
    // GtkWidget* button;
    // button = gtk_button_new_with_label("Stop");
    // gtk_container_add(GTK_CONTAINER(window),button);
    // int state = 1;
    // gpointer ptr = GINT_TO_POINTER(state);
    // g_signal_connect(button,"clicked",G_CALLBACK(clicked_button),ptr);
    // gtk_widget_show_all(window);
    // gtk_main();
    // // sleep(1);

    // int i = 0;
    // while (state){
    //     std::cout << i << std::endl;
    //     i += 1;
    // }

    // free(UIntimage1);
    // free(UIntimage2);
    // free(fimg1);
    // free(fimg2);
    hipDeviceReset();
    return 0;
}