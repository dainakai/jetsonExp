#include "CUDAfunctions.cuh"
#include "hostfunctions.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "Spinnaker.h"
#include "SpinGenApi/SpinnakerGenApi.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core/types_c.h>

int main(int argc, char** argv){
    printf("%s Starting...\n", argv[0]);

    const char* pathImg1 = "./cam1.bmp";
    const char* pathImg2 = "./cam2.bmp";

    const int imgLen = 1024;
    const int intrSize = imgLen/8;
    const int gridSize = imgLen/8;
    const int srchSize = imgLen/4;
    const int gridNum = (int)(imgLen/gridSize);

    const int blockSize = 16;

    unsigned char *UIntimage1, *UIntimage2;
    float *fimg1, *fimg2;
    UIntimage1 = (unsigned char*)malloc(sizeof(unsigned char)*imgLen*imgLen);
    UIntimage2 = (unsigned char*)malloc(sizeof(unsigned char)*imgLen*imgLen);
    fimg1 = (float*)malloc(sizeof(float)*imgLen*imgLen);
    fimg2 = (float*)malloc(sizeof(float)*imgLen*imgLen);

    getFloatimage(fimg1,imgLen,pathImg1);
    getFloatimage(fimg2,imgLen,pathImg2);

    float vecArrayX[(gridNum-1)*(gridNum-1)];
    float vecArrayY[(gridNum-1)*(gridNum-1)];
    float *pvecArrX = (float *)vecArrayX;
    float *pvecArrY = (float *)vecArrayY;
    getPIVMapOnGPU(pvecArrX,pvecArrY,fimg1,fimg2,imgLen,gridSize,intrSize,srchSize,blockSize);

    for (int i = 0; i < gridNum-1; i++)
    {
        for (int j = 0; j < gridNum-1; j++)
        {
            printf("vx: %f\t vy: %f\n",vecArrayX[i*(gridNum-1) +j],vecArrayY[i*(gridNum-1) +j]);
        }
        printf("\n");
    }

    saveVecArray(vecArrayX,vecArrayY,gridSize,gridNum);
    plotVecFieldOnGnuplot(imgLen);

    Spinnaker::SystemPtr system = Spinnaker::System::GetInstance();
    Spinnaker::CameraList camList = system->GetCameras();
    cameraSetup(camList,1024,600,400);

    system->ReleaseInstance();

    free(UIntimage1);
    free(UIntimage2);
    free(fimg1);
    free(fimg2);
    return 0;
}